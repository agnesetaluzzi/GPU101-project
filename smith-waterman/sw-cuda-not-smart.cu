
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>

#define S_LEN 64
#define N 1000

// penalties
#define ins -2
#define del -2
#define match 1
#define mismatch -1

// error handling for CUDA API functions
#define CHECK(call)                                                  \
    {                                                                \
        const hipError_t err = call;                                \
        if (err != hipSuccess)                                      \
        {                                                            \
            printf("%s in %s at line %d\n", hipGetErrorString(err), \
                   __FILE__, __LINE__);                              \
            exit(EXIT_FAILURE);                                      \
        }                                                            \
    }

// error handling for kernel invocations
#define CHECK_KERNELCALL()                                           \
    {                                                                \
        const hipError_t err = hipGetLastError();                  \
        if (err != hipSuccess)                                      \
        {                                                            \
            printf("%s in %s at line %d\n", hipGetErrorString(err), \
                   __FILE__, __LINE__);                              \
            exit(EXIT_FAILURE);                                      \
        }                                                            \
    }

double get_time() // function to get the time of day in seconds
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + tv.tv_usec * 1e-6;
}

int max4(int n1, int n2, int n3, int n4)
{
    int tmp1, tmp2;
    tmp1 = n1 > n2 ? n1 : n2;
    tmp2 = n3 > n4 ? n3 : n4;
    tmp1 = tmp1 > tmp2 ? tmp1 : tmp2;
    return tmp1;
}

void backtrace(char *simple_rev_cigar, char **dir_mat, int i, int j, int max_cigar_len)
{
    int n;
    for (n = 0; n < max_cigar_len && dir_mat[i][j] != 0; n++)
    {
        int dir = dir_mat[i][j];
        if (dir == 1 || dir == 2)
        {
            i--;
            j--;
        }
        else if (dir == 3)
            i--;
        else if (dir == 4)
            j--;

        simple_rev_cigar[n] = dir;
    }
}

__global__ void kernel_main(char *d_query, char *d_reference, int *d_res, char *d_simple_rev_cigar)
{
    // int index = blockIdx.x * blockDim.x + threadIdx.x; // index of each element of the array
    __shared__ int d_sc_mat[S_LEN + 1][S_LEN + 1];
    __shared__ char d_dir_mat[S_LEN + 1][S_LEN + 1];
    int max = ins; // in sw all scores of the alignment are >= 0, so this will be for sure changed
    int maxi, maxj;
    // initialize the scoring matrix and direction matrix to 0
    for (int i = 0; i < S_LEN + 1; i++)
    {
        for (int j = 0; j < S_LEN + 1; j++)
        {
            d_sc_mat[i][j] = {0};
            d_dir_mat[i][j] = {0};
        }
    }

    for (int i = 1; i < S_LEN + 1; i++)
    {
        for (int j = 1; j < S_LEN + 1; j++)
        {
            // compare the sequences characters
            int comparison = (d_query[blockIdx.x * S_LEN + i - 1] == d_reference[blockIdx.x * S_LEN + j - 1]) ? match : mismatch;
            // compute the cell knowing the comparison result
            int tmp1, tmp2;
            tmp1 = (d_sc_mat[i - 1][j - 1] + comparison) > (d_sc_mat[i - 1][j] + del) ? (d_sc_mat[i - 1][j - 1] + comparison) : (d_sc_mat[i - 1][j] + del);
            tmp2 = (d_sc_mat[i][j - 1] + ins) > 0 ? (d_sc_mat[i][j - 1] + ins) : 0;
            int tmp = tmp1 > tmp2 ? tmp1 : tmp2;
            char dir;

            if (tmp == (d_sc_mat[i - 1][j - 1] + comparison))
                dir = comparison == match ? 1 : 2;
            else if (tmp == (d_sc_mat[i - 1][j] + del))
                dir = 3;
            else if (tmp == (d_sc_mat[i][j - 1] + ins))
                dir = 4;
            else
                dir = 0;

            d_dir_mat[i][j] = dir;
            d_sc_mat[i][j] = tmp;

            if (tmp > max)
            {
                max = tmp;
                maxi = i;
                maxj = j;
            }
        }
    }

    d_res[blockIdx.x] = d_sc_mat[maxi][maxj];

    int i = maxi;
    int j = maxj;
    for (int n = 0; n < S_LEN * 2 && d_dir_mat[i][j] != 0; n++)
    {
        int dir = d_dir_mat[i][j];
        if (dir == 1 || dir == 2)
        {
            i--;
            j--;
        }
        else if (dir == 3)
            i--;
        else if (dir == 4)
            j--;

        d_simple_rev_cigar[blockIdx.x * 2 * S_LEN + n] = dir;
    }
}

int main(int argc, char *argv[])
{
    srand(time(NULL));

    char alphabet[5] = {'A', 'C', 'G', 'T', 'N'};

    // host memory allocation
    char **query = (char **)malloc(N * sizeof(char *));
    for (int i = 0; i < N; i++)
        query[i] = (char *)malloc(S_LEN * sizeof(char));
    char *query_copy = (char *)malloc(N * S_LEN * sizeof(char));

    char **reference = (char **)malloc(N * sizeof(char *));
    for (int i = 0; i < N; i++)
        reference[i] = (char *)malloc(S_LEN * sizeof(char));
    char *reference_copy = (char *)malloc(N * S_LEN * sizeof(char));

    int **sc_mat = (int **)malloc((S_LEN + 1) * sizeof(int *));
    for (int i = 0; i < (S_LEN + 1); i++)
        sc_mat[i] = (int *)malloc((S_LEN + 1) * sizeof(int));
    char **dir_mat = (char **)malloc((S_LEN + 1) * sizeof(char *));
    for (int i = 0; i < (S_LEN + 1); i++)
        dir_mat[i] = (char *)malloc((S_LEN + 1) * sizeof(char));

    int *res = (int *)malloc(N * sizeof(int));
    int *res_gpu = (int *)malloc(N * sizeof(int));
    char **simple_rev_cigar = (char **)malloc(N * sizeof(char *));
    for (int i = 0; i < N; i++)
        simple_rev_cigar[i] = (char *)malloc(S_LEN * 2 * sizeof(char));
    char *simple_rev_cigar_gpu = (char *)malloc(N * S_LEN * 2 * sizeof(char));

    // randomly generate sequences
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < S_LEN; j++)
        {
            query[i][j] = alphabet[rand() % 5];
            query_copy[i * S_LEN + j] = query[i][j];

            reference[i][j] = alphabet[rand() % 5];
            reference_copy[i * S_LEN + j] = reference[i][j];
        }
    }

    // device memory allocation
    char *d_query, *d_reference, *d_simple_rev_cigar;
    int *d_res;

    CHECK(hipMalloc(&d_query, N * S_LEN * sizeof(char)));

    CHECK(hipMalloc(&d_reference, N * S_LEN * sizeof(char)));

    CHECK(hipMalloc(&d_res, N * sizeof(int)));
    CHECK(hipMalloc(&d_simple_rev_cigar, N * S_LEN * 2 * sizeof(char)));

    // CPU->GPU data transmission
    CHECK(hipMemcpy(d_query, query_copy, sizeof(char) * N * S_LEN, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_reference, reference_copy, sizeof(char) * N * S_LEN, hipMemcpyHostToDevice));

    // CPU execution
    double start_cpu = get_time();

    for (int n = 0; n < N; n++)
    {
        int max = ins; // in sw all scores of the alignment are >= 0, so this will be for sure changed
        int maxi, maxj;
        // initialize the scoring matrix and direction matrix to 0
        for (int i = 0; i < S_LEN + 1; i++)
        {
            for (int j = 0; j < S_LEN + 1; j++)
            {
                sc_mat[i][j] = 0;
                dir_mat[i][j] = 0;
            }
        }
        // compute the alignment
        for (int i = 1; i < S_LEN + 1; i++)
        {
            for (int j = 1; j < S_LEN + 1; j++)
            {
                // compare the sequences characters
                int comparison = (query[n][i - 1] == reference[n][j - 1]) ? match : mismatch;
                // compute the cell knowing the comparison result
                int tmp = max4(sc_mat[i - 1][j - 1] + comparison, sc_mat[i - 1][j] + del, sc_mat[i][j - 1] + ins, 0);
                char dir;

                if (tmp == (sc_mat[i - 1][j - 1] + comparison))
                    dir = comparison == match ? 1 : 2;
                else if (tmp == (sc_mat[i - 1][j] + del))
                    dir = 3;
                else if (tmp == (sc_mat[i][j - 1] + ins))
                    dir = 4;
                else
                    dir = 0;

                dir_mat[i][j] = dir;
                sc_mat[i][j] = tmp;

                if (tmp > max)
                {
                    max = tmp;
                    maxi = i;
                    maxj = j;
                }
            }
        }
        res[n] = sc_mat[maxi][maxj];
        backtrace(simple_rev_cigar[n], dir_mat, maxi, maxj, S_LEN * 2);
    }

    double end_cpu = get_time();

    // GPU execution
    double start_gpu = get_time();

    dim3 blocksPerGrid(N, 1, 1);
    dim3 threadsPerBlock(1, 1, 1);

    kernel_main<<<blocksPerGrid, threadsPerBlock>>>(d_query, d_reference, d_res, d_simple_rev_cigar);
    CHECK_KERNELCALL();

    CHECK(hipDeviceSynchronize());

    double end_gpu = get_time();

    CHECK(hipMemcpy(res_gpu, d_res, sizeof(int) * N, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(simple_rev_cigar_gpu, d_simple_rev_cigar, sizeof(char) * N * S_LEN * 2, hipMemcpyDeviceToHost));

    for (int n = 0; n < N; n++)
    {
        if (res[n] != res_gpu[n])
        {
            fprintf(stderr, "ERRORE, RISULTATO SBAGLIATO SU GPU\n");
            break;
        }
        for (int s = 0; s < S_LEN * 2; s++)
        {
            if (simple_rev_cigar[n][s] != simple_rev_cigar_gpu[n * S_LEN * 2 + s])
            {
                fprintf(stderr, "ERRORE, RISULTATO SBAGLIATO SU GPU (BACKTRACE)\n");
                break;
            }
        }
    }

    printf("SW Time CPU: %.10lf\n", end_cpu - start_cpu);
    printf("SW Time GPU: %.10lf\n", end_gpu - start_gpu);

    CHECK(hipFree(d_query));
    CHECK(hipFree(d_reference));
    CHECK(hipFree(d_res));
    CHECK(hipFree(d_simple_rev_cigar));

    free(query);
    free(query_copy);
    free(reference);
    free(reference_copy);
    free(sc_mat);
    free(dir_mat);
    free(res);
    free(res_gpu);
    free(simple_rev_cigar);

    return 0;
}