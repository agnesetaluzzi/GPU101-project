
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>

#define S_LEN 512
#define N 1000

// penalties
#define ins -2
#define del -2
#define match 1
#define mismatch -1

// error handling for CUDA API functions
#define CHECK(call)                                                  \
    {                                                                \
        const hipError_t err = call;                                \
        if (err != hipSuccess)                                      \
        {                                                            \
            printf("%s in %s at line %d\n", hipGetErrorString(err), \
                   __FILE__, __LINE__);                              \
            exit(EXIT_FAILURE);                                      \
        }                                                            \
    }

// error handling for kernel invocations
#define CHECK_KERNELCALL()                                           \
    {                                                                \
        const hipError_t err = hipGetLastError();                  \
        if (err != hipSuccess)                                      \
        {                                                            \
            printf("%s in %s at line %d\n", hipGetErrorString(err), \
                   __FILE__, __LINE__);                              \
            exit(EXIT_FAILURE);                                      \
        }                                                            \
    }

double get_time() // function to get the time of day in seconds
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + tv.tv_usec * 1e-6;
}

int max4(int n1, int n2, int n3, int n4)
{
    int tmp1, tmp2;
    tmp1 = n1 > n2 ? n1 : n2;
    tmp2 = n3 > n4 ? n3 : n4;
    tmp1 = tmp1 > tmp2 ? tmp1 : tmp2;
    return tmp1;
}

void backtrace(char *simple_rev_cigar, char **dir_mat, int i, int j, int max_cigar_len)
{
    int n;
    for (n = 0; n < max_cigar_len && dir_mat[i][j] != 0; n++)
    {
        int dir = dir_mat[i][j];
        if (dir == 1 || dir == 2)
        {
            i--;
            j--;
        }
        else if (dir == 3)
            i--;
        else if (dir == 4)
            j--;

        simple_rev_cigar[n] = dir;
    }
}

void sw(int **sc_mat, char **dir_mat, char **query, char **reference, int *res, char **simple_rev_cigar)
{
    for (int n = 0; n < N; n++)
    {
        int max = ins; // in sw all scores of the alignment are >= 0, so this will be for sure changed
        int maxi, maxj;
        // initialize the scoring matrix and direction matrix to 0
        for (int i = 0; i < S_LEN + 1; i++)
        {
            for (int j = 0; j < S_LEN + 1; j++)
            {
                sc_mat[i][j] = 0;
                dir_mat[i][j] = 0;
            }
        }
        // compute the alignment
        for (int i = 1; i < S_LEN + 1; i++)
        {
            for (int j = 1; j < S_LEN + 1; j++)
            {
                // compare the sequences characters
                int comparison = (query[n][i - 1] == reference[n][j - 1]) ? match : mismatch;
                // compute the cell knowing the comparison result
                int tmp = max4(sc_mat[i - 1][j - 1] + comparison, sc_mat[i - 1][j] + del, sc_mat[i][j - 1] + ins, 0);
                char dir;

                if (tmp == (sc_mat[i - 1][j - 1] + comparison))
                    dir = comparison == match ? 1 : 2;
                else if (tmp == (sc_mat[i - 1][j] + del))
                    dir = 3;
                else if (tmp == (sc_mat[i][j - 1] + ins))
                    dir = 4;
                else
                    dir = 0;

                dir_mat[i][j] = dir;
                sc_mat[i][j] = tmp;

                if (tmp > max)
                {
                    max = tmp;
                    maxi = i;
                    maxj = j;
                }
            }
        }
        res[n] = sc_mat[maxi][maxj];
        backtrace(simple_rev_cigar[n], dir_mat, maxi, maxj, S_LEN * 2);
    }
}

__global__ void collect_res(int *d_res)
{
    int threadId = threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadId;

    for (int i = blockDim.x / 2; i > 0; i >>= 1)
    {
        if (threadId < i)
        {
            d_res[index] = d_res[index] > d_res[index + i] ? d_res[index] : d_res[index + i];
        }
        __syncthreads();
    }
}

__global__ void sw_gpu(char *d_query, char *d_reference, int *d_sc_mat, char *d_dir_mat, int *d_res)
{
    int threadId = threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadId; // index of each element of the array

    // __shared__ char d_dir_mat[S_LEN + 1][S_LEN + 1];

    int max = ins; // in sw all scores of the alignment are >= 0, so this will be for sure changed
    int maxi, maxj;

    // initialize the scoring matrix and direction matrix to 0
    int blockShift = blockIdx.x * (S_LEN + 1) * (S_LEN + 1);
    for (int j = 0; j < S_LEN + 1; j++)
    {
        d_sc_mat[blockShift + threadId * (S_LEN + 1) + j] = {0};
        d_dir_mat[blockShift + threadId * (S_LEN + 1) + j] = {0};
        if (threadId == 0)
        {
            d_sc_mat[blockShift + S_LEN * (S_LEN + 1) + j] = {0};
            d_dir_mat[blockShift + S_LEN * (S_LEN + 1) + j] = {0};
        }
    }
    __syncthreads();

    for (int d = 0; d < S_LEN * 2 + 1; d++)
    {
        // calculate the indexes
        int i = threadId + 1;
        int j = d - threadId + 1;

        // check if valid
        if (i >= 1 && i <= S_LEN && j >= 1 && j <= S_LEN)
        {

            // compare the sequences characters
            int comparison = (d_query[blockIdx.x * blockDim.x + i - 1] == d_reference[blockIdx.x * blockDim.x + j - 1]) ? match : mismatch;

            // compute the cell knowing the comparison result
            int tmp1, tmp2;
            tmp1 = (d_sc_mat[blockShift + (i - 1) * (S_LEN + 1) + j - 1] + comparison) > (d_sc_mat[blockShift + (i - 1) * (S_LEN + 1) + j] + del) ? (d_sc_mat[blockShift + (i - 1) * (S_LEN + 1) + j - 1] + comparison) : (d_sc_mat[blockShift + (i - 1) * (S_LEN + 1) + j] + del);
            tmp2 = (d_sc_mat[blockShift + i * (S_LEN + 1) + j - 1] + ins) > 0 ? (d_sc_mat[blockShift + i * (S_LEN + 1) + j - 1] + ins) : 0;
            int tmp = tmp1 > tmp2 ? tmp1 : tmp2;
            char dir;

            if (tmp == (d_sc_mat[blockShift + (i - 1) * (S_LEN + 1) + j - 1] + comparison))
                dir = comparison == match ? 1 : 2;
            else if (tmp == (d_sc_mat[blockShift + (i - 1) * (S_LEN + 1) + j] + del))
                dir = 3;
            else if (tmp == (d_sc_mat[blockShift + i * (S_LEN + 1) + j - 1] + ins))
                dir = 4;
            else
                dir = 0;

            d_dir_mat[blockShift + i * (S_LEN + 1) + j] = dir;
            d_sc_mat[blockShift + i * (S_LEN + 1) + j] = tmp;

            if (tmp > max)
            {
                max = tmp;
                maxi = i;
                maxj = j;
            }
        }
        __syncthreads();
    }

    d_res[index] = max;
    // d_maxi_temp[index] = maxi;
    // d_maxj_temp[index] = maxj;
}

int main(int argc, char *argv[])
{
    srand(time(NULL));

    char alphabet[5] = {'A', 'C', 'G', 'T', 'N'};

    // host memory allocation
    char **query = (char **)malloc(N * sizeof(char *));
    for (int i = 0; i < N; i++)
        query[i] = (char *)malloc(S_LEN * sizeof(char));
    char *query_copy = (char *)malloc(N * S_LEN * sizeof(char));

    char **reference = (char **)malloc(N * sizeof(char *));
    for (int i = 0; i < N; i++)
        reference[i] = (char *)malloc(S_LEN * sizeof(char));
    char *reference_copy = (char *)malloc(N * S_LEN * sizeof(char));

    int **sc_mat = (int **)malloc((S_LEN + 1) * sizeof(int *));
    for (int i = 0; i < (S_LEN + 1); i++)
        sc_mat[i] = (int *)malloc((S_LEN + 1) * sizeof(int));
    char **dir_mat = (char **)malloc((S_LEN + 1) * sizeof(char *));
    for (int i = 0; i < (S_LEN + 1); i++)
        dir_mat[i] = (char *)malloc((S_LEN + 1) * sizeof(char));

    int *res = (int *)malloc(N * sizeof(int));
    int *res_gpu = (int *)malloc(N * S_LEN * sizeof(int));

    char **simple_rev_cigar = (char **)malloc(N * sizeof(char *));
    for (int i = 0; i < N; i++)
        simple_rev_cigar[i] = (char *)malloc(S_LEN * 2 * sizeof(char));
    char *simple_rev_cigar_gpu = (char *)malloc(N * S_LEN * 2 * sizeof(char));

    // randomly generate sequences
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < S_LEN; j++)
        {
            query[i][j] = alphabet[rand() % 5];
            query_copy[i * S_LEN + j] = query[i][j];

            reference[i][j] = alphabet[rand() % 5];
            reference_copy[i * S_LEN + j] = reference[i][j];
        }
    }

    // device memory allocation
    char *d_query, *d_reference, *d_dir_mat, *d_simple_rev_cigar;
    int *d_res, *d_sc_mat;
    // int *d_maxi_temp, *d_maxj_temp;

    CHECK(hipMalloc(&d_query, N * S_LEN * sizeof(char)));
    CHECK(hipMalloc(&d_reference, N * S_LEN * sizeof(char)));

    CHECK(hipMalloc(&d_sc_mat, N * (S_LEN + 1) * (S_LEN + 1) * sizeof(int)));
    CHECK(hipMalloc(&d_dir_mat, N * (S_LEN + 1) * (S_LEN + 1) * sizeof(char)));

    CHECK(hipMalloc(&d_res, N * S_LEN * sizeof(int)));
    // CHECK(cudaMalloc(&d_maxi_temp, N * S_LEN * sizeof(int)));
    // CHECK(cudaMalloc(&d_maxj_temp, N * S_LEN * sizeof(int)));

    CHECK(hipMalloc(&d_simple_rev_cigar, N * S_LEN * 2 * sizeof(char)));

    // CPU->GPU data transmission
    CHECK(hipMemcpy(d_query, query_copy, sizeof(char) * N * S_LEN, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_reference, reference_copy, sizeof(char) * N * S_LEN, hipMemcpyHostToDevice));

    // CPU execution
    double start_cpu = get_time();
    sw(sc_mat, dir_mat, query, reference, res, simple_rev_cigar);
    double end_cpu = get_time();

    // GPU execution
    double start_gpu = get_time();
    dim3 blocksPerGrid(N, 1, 1);
    dim3 threadsPerBlock(S_LEN, 1, 1);
    sw_gpu<<<blocksPerGrid, threadsPerBlock>>>(d_query, d_reference, d_sc_mat, d_dir_mat, d_res);
    CHECK_KERNELCALL();
    CHECK(hipDeviceSynchronize());
    collect_res<<<blocksPerGrid, threadsPerBlock>>>(d_res);
    CHECK_KERNELCALL();
    CHECK(hipDeviceSynchronize());
    double end_gpu = get_time();

    CHECK(hipMemcpy(res_gpu, d_res, sizeof(int) * N * S_LEN, hipMemcpyDeviceToHost));
    // CHECK(cudaMemcpy(simple_rev_cigar_gpu, d_simple_rev_cigar, sizeof(char) * N * S_LEN * 2, cudaMemcpyDeviceToHost));

    for (int n = 0; n < N; n++)
    {
        // printf("correct: %d, mine: %d\n", res[n], res_gpu[n]);
        if (res[n] != res_gpu[n * S_LEN])
        {
            fprintf(stderr, "ERRORE, RISULTATO SBAGLIATO SU GPU\n");
            break;
        }
        /*for (int s = 0; s < S_LEN * 2; s++)
        {
            if (simple_rev_cigar[n][s] != simple_rev_cigar_gpu[n * S_LEN * 2 + s])
            {
                fprintf(stderr, "ERRORE, RISULTATO SBAGLIATO SU GPU (BACKTRACE)\n");
                break;
            }
        }*/
    }

    printf("SW Time CPU: %.10lf\n", end_cpu - start_cpu);
    printf("SW Time GPU: %.10lf\n", end_gpu - start_gpu);

    CHECK(hipFree(d_query));
    CHECK(hipFree(d_reference));
    CHECK(hipFree(d_sc_mat));
    CHECK(hipFree(d_dir_mat));
    CHECK(hipFree(d_res));
    // CHECK(cudaFree(d_maxi_temp));
    // CHECK(cudaFree(d_maxj_temp));
    CHECK(hipFree(d_simple_rev_cigar));

    free(query);
    free(query_copy);
    free(reference);
    free(reference_copy);
    free(sc_mat);
    free(dir_mat);
    free(res);
    free(res_gpu);
    free(simple_rev_cigar);

    return 0;
}